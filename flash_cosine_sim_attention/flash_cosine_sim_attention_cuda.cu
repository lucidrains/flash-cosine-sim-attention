#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <c10/cuda/CUDAGuard.h>

#include <torch/extension.h>

// error handler
// from https://leimao.github.io/blog/Proper-CUDA-Error-Checking

#define CHECK_LAST_CUDA_ERROR() check(__FILE__, __LINE__)
void check(const char* file, const int line)
{
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        std::cerr << "CUDA Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
    }
}

#define ACCESSOR(x, n, type) x.packed_accessor32<type, n, torch::RestrictPtrTraits>()

// type alias

template <typename scalar_t, int dims>
using PackedAccessor = torch::PackedTensorAccessor32<scalar_t, dims, torch::RestrictPtrTraits>;

// helper functions

__host__ __device__ int cdiv(int numer, int denom) {
    return (numer + denom - 1) / denom;
}

__host__ __device__ int next_multiple_of(int num, int multiple_of) {
    return cdiv(num, multiple_of) * multiple_of;
}

__host__ __device__ int next_pow_2(int n) {
    int i = 1;
    while(i < n)
        i *= 2;
    return i;
}

__device__ void warp_reduce(volatile float* sm, int tid, int max) {
    for (int s = 32; s > 0; s>>=1) {
        if ((tid + s) >= max)
            continue;

        sm[tid] += sm[tid + s];
    }
}

bool divisible_by(int num, int denom) {
    return (num % denom) == 0;
}

// mma and smem fragment

// mma

struct mma_warp_tile {
    // How much data is processed by a single thread:
    static constexpr int N_thread = 4;
    static constexpr int M_thread = 4;

    // Thread layout within a warp:
    static constexpr int N_warp = 8;
    static constexpr int M_warp = 4;
    static_assert(N_warp * M_warp == 32);

    // Warp layout within a block:
    static constexpr int N_block = 2;
    static constexpr int M_block = 4;
    static_assert(N_block * M_block * N_warp * M_warp == 256); // blockDim.x

    // Dimensions of the tile, in threads:
    static constexpr int N_tile = N_warp * N_block * N_thread;
    static constexpr int M_tile = M_warp * M_block * M_thread;

    static constexpr float IS_NULL_FLOAT = -3.14159e6;
    static constexpr float MASK_VALUE = -1e8;

    // Registers:
    float A_frag[N_thread];            // N x 1 fragment
    float B_frag[M_thread];            // 1 x M fragment
    float C_frag[N_thread * M_thread]; // N x M fragment


    int warp_x;   // x offset of the warp within the block tile
    int warp_y;   // y offset of the warp within the block tile
    int thread_x; // x offset of the thread within the warp tile
    int thread_y; // y offset of the thread within the warp tile

    __device__ mma_warp_tile() {
        int warp_id = threadIdx.x / 32;
        warp_x = (warp_id % M_block);
        warp_y = (warp_id / M_block);

        int lane_id = threadIdx.x % 32;
        thread_x = warp_x * M_warp * M_thread + lane_id % M_warp;
        thread_y = warp_y * N_warp * N_thread + lane_id / M_warp;
    }

    // Initialize C to all zeros
    __device__ void zero() {
        for (int i = 0; i < N_thread * M_thread; i++) {
            C_frag[i] = 0.f;
        }
    }

    // Performs C = A * B + C
    __device__ void mma(
        const float* A_sm_ptr,
        const float* B_sm_ptr,
        int k,
        bool has_mask,
        const float is_null_float
    ) {
        // Load a N x 1 fragment of A from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            A_frag[i] = A_sm_ptr[i * N_warp + thread_y + k * N_tile];
        }

        // Load a 1 x M fragment of B from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < M_thread; i++) {
            B_frag[i] = B_sm_ptr[i * M_warp + thread_x + k * M_tile];
        }

        // Compute:
        #pragma unroll
        for (int j = 0; j < M_thread ; j++) {

            bool is_masked_out = false;
            if (has_mask) {
                is_masked_out = B_sm_ptr[j * M_warp + thread_x] == is_null_float;
            }

            #pragma unroll
            for (int i = 0; i < N_thread; i++) {
                if (is_masked_out) {
                    C_frag[i * M_thread + j] = MASK_VALUE;
                } else {
                    C_frag[i * M_thread + j] += A_frag[i] * B_frag[j];
                }
            }
        }
    }

    // Perform a pointwise operation, specified by the given lambda, on C
    template<typename F>
    __device__ void pointwise(F&& op) {
        #pragma unroll
        for (int i = 0; i < N_thread * M_thread; i++) {
            C_frag[i] = op(C_frag[i], i);
        }
    }

    // Copy C from registers to shared memory
    __device__ void store(float* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[(thread_y + i * N_warp) * M_tile + j * M_warp + thread_x]
                  = C_frag[i * M_thread + j];
            }
        }
    }

    __device__ void store_transpose(float* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[thread_y + i * N_warp + (j * M_warp + thread_x) * N_tile]
                  = C_frag[i * M_thread + j];
            }
        }
    }
};


struct out_mma_warp_tile {
    // How much data is processed by a single thread:
    static constexpr int N_thread = 4;
    static constexpr int M_thread = 4;

    // Thread layout within a warp:
    static constexpr int N_warp = 8;
    static constexpr int M_warp = 4;
    static_assert(N_warp * M_warp == 32);

    // Warp layout within a block:
    static constexpr int N_block = 2;
    static constexpr int M_block = 4;
    static_assert(N_block * M_block * N_warp * M_warp == 256); // blockDim.x

    // Dimensions of the tile, in threads:
    static constexpr int N_tile = N_warp * N_block * N_thread;
    static constexpr int M_tile = M_warp * M_block * M_thread;

    static constexpr float EPS = 1e-10;

    // Registers:
    float A_frag[N_thread];            // N x 1 fragment
    float B_frag[M_thread];            // 1 x M fragment
    float L_frag[N_thread];            // N x 1 fragment
    float C_frag[N_thread * M_thread]; // N x M fragment

    int warp_x;   // x offset of the warp within the block tile
    int warp_y;   // y offset of the warp within the block tile
    int thread_x; // x offset of the thread within the warp tile
    int thread_y; // y offset of the thread within the warp tile

    __device__ out_mma_warp_tile() {
        int warp_id = threadIdx.x / 32;
        warp_x = (warp_id % M_block);
        warp_y = (warp_id / M_block);

        int lane_id = threadIdx.x % 32;
        thread_x = warp_x * M_warp * M_thread + lane_id % M_warp;
        thread_y = warp_y * N_warp * N_thread + lane_id / M_warp;
    }

    // Initialize C to all zeros
    __device__ void zero() {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            L_frag[i] = 0.f;
        }

        #pragma unroll
        for (int i = 0; i < N_thread * M_thread; i++) {
            C_frag[i] = 0.f;
        }
    }

    // Performs C = A * B + C
    __device__ void mma(
        const float* A_sm_ptr,
        const float* B_sm_ptr,
        int k
    ) {
        // Load a N x 1 fragment of A from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            A_frag[i] = A_sm_ptr[i * N_warp + thread_y + k * N_tile];
        }

        // Load a 1 x M fragment of B from shared memory to registers:
        #pragma unroll
        for (int i = 0; i < M_thread; i++) {
            B_frag[i] = B_sm_ptr[i * M_warp + thread_x + k * M_tile];
        }

        // Compute:
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            L_frag[i] += A_frag[i];

            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_frag[i * M_thread + j] += A_frag[i] * B_frag[j];
            }
        }
    }

    // Perform a pointwise operation, specified by the given lambda, on C
    template<typename F>
    __device__ void pointwise(F&& op) {
        #pragma unroll
        for (int i = 0; i < N_thread * M_thread; i++) {
            C_frag[i] = op(C_frag[i], i);
        }
    }

    // Copy C from registers to shared memory
    __device__ void store(float* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[(thread_y + i * N_warp) * M_tile + j * M_warp + thread_x]
                  = C_frag[i * M_thread + j] / max(L_frag[i], EPS);
            }
        }
    }

    template<typename accessor>
    __device__ void store_rowsum(accessor gmem, int tile_y, int max_y) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            int gmem_idx = tile_y * N_tile + i * N_warp + thread_y;

            if (gmem_idx >= max_y)
                continue;

            gmem[gmem_idx] = L_frag[i];
        }
    }

    __device__ void store_transpose(float* C_sm_ptr) {
        #pragma unroll
        for (int i = 0; i < N_thread; i++) {
            #pragma unroll
            for (int j = 0; j < M_thread ; j++) {
                C_sm_ptr[thread_y + i * N_warp + (j * M_warp + thread_x) * N_tile]
                  = C_frag[i * M_thread + j] / max(L_frag[i], EPS);
            }
        }
    }
};
// shared memory fragment

template<typename T>
struct smem_fragment {
    T* smem;
    int N;
    int M;

    __device__ smem_fragment(T* shared_base, int N, int M)
      : smem(shared_base), N(N), M(M) { }

    __device__ void load(const T* gmem) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            smem[i] = gmem[i];
        }
    }

    template<typename accessor>
    __device__ void load(accessor gmem, int tile_x, int tile_y, int max_y) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int x = i % M;
            int y = i / M;
            int gmem_y = y + tile_y * N;
            int gmem_x = x + tile_x * M;

            if (gmem_y >= max_y)
                continue;

            smem[i] = gmem[gmem_y][gmem_x];
        }
    }

    template<typename accessor>
    __device__ void load_transpose(accessor gmem, int tile_x, int tile_y, int max_y) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int y = i % M;
            int x = i / M;
            int gmem_y = x + tile_y * N;
            int gmem_x = y + tile_x * M;

            if (gmem_y >= max_y)
                continue;

            smem[y * N + x] = gmem[gmem_y][gmem_x];
        }
    }

    template<typename accessor, typename accessor_mask>
    __device__ void load_transpose(accessor gmem, int tile_x, int tile_y, bool has_mask, accessor_mask mask, const float is_null_float, int max_y) {
        if (!has_mask)
            return load_transpose(gmem, tile_x, tile_y, max_y);

        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int y = i % M;
            int x = i / M;
            int gmem_y = x + tile_y * N;
            int gmem_x = y + tile_x * M;

            if (y == 0 && !mask[gmem_y]) {
                smem[y * N + x] = is_null_float;
                continue;
            }

            if (gmem_y >= max_y)
                continue;

            smem[y * N + x] = gmem[gmem_y][gmem_x];
        }
    }

    template<typename accessor>
    __device__ void store(accessor gmem, int tile_x, int tile_y, int max_y) {
        for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
            int x = i % M;
            int y = i / M;
            int gmem_y = y + tile_y * N;
            int gmem_x = x + tile_x * M;

            if (gmem_y >= max_y) {
                continue;
            }

            gmem[gmem_y][gmem_x] = smem[i];
        }
    }

    __device__ unsigned size() {
        return N * M;
    }

    __device__ T* next() {
        return smem + size();
    }
};

// forward kernel

__global__ void forward_kernel(
    const PackedAccessor<float, 4> Q,
    const PackedAccessor<float, 4> K,
    const PackedAccessor<float, 4> V,
          PackedAccessor<float, 4> O,
          PackedAccessor<float, 3> L,
    const PackedAccessor<bool, 2> mask,
    const PackedAccessor<float, 3> attn_bias,
    const float scale,
    const bool causal,
    const bool has_mask,
    const bool has_attn_bias
) {
    const int H = Q.size(1);
    const int N = Q.size(2);
    const int M = K.size(2);
    const int D = Q.size(3);
    const int E = V.size(3);

    const int MN_DIFF = M - N;  // for calculating causality when query and key lengths differ

    const int batch = blockIdx.y / H;
    const int heads = blockIdx.y % H;

    // shortcut accessor

    auto Q_ = Q[batch][heads];
    auto K_ = K[batch][heads];
    auto V_ = V[batch][heads];
    auto O_ = O[batch][heads];
    auto L_ = L[batch][heads];
    auto attn_bias_ = attn_bias[heads];

    // tiles

    const int tile_w = cdiv(M, mma_warp_tile::M_tile);
    const int tile_y = blockIdx.x;

    // shared memory

    extern __shared__ float _shared_mem[];

    mma_warp_tile QK_mma; // 32x16 tile per warp in registers -> process 64x64 with the block
    out_mma_warp_tile out_mma;

    smem_fragment<float> Q_sm{_shared_mem, mma_warp_tile::N_tile, D};
    smem_fragment<float> O_sm{_shared_mem, mma_warp_tile::N_tile, E};
    smem_fragment<float> A_sm{(E > D ? O_sm.next() : A_sm.next()), mma_warp_tile::N_tile, mma_warp_tile::M_tile};
    smem_fragment<float> K_sm{A_sm.next(), mma_warp_tile::M_tile, D};
    smem_fragment<float> V_sm{A_sm.next(), mma_warp_tile::M_tile, E};

    // helper variables

    int global_row, global_col;
    float bias;

    // start loop

    out_mma.zero();

    Q_sm.load_transpose(Q_, 0, tile_y, N);

    for (int tile_x = 0; tile_x < tile_w; tile_x++) {
        if (causal && (mma_warp_tile::M_tile * tile_x - MN_DIFF) >= (mma_warp_tile::N_tile * (tile_y + 1)))
            continue;

        K_sm.load_transpose(K_, 0, tile_x, has_mask, mask[batch], mma_warp_tile::IS_NULL_FLOAT, M);

        __syncthreads();

        QK_mma.zero();

        for (int d = 0; d < D; d++) {
            QK_mma.mma(Q_sm.smem, K_sm.smem, d, has_mask, mma_warp_tile::IS_NULL_FLOAT);
        }

        QK_mma.pointwise([&](float el, int index) {
            global_row = tile_y * mma_warp_tile::N_tile + (index / mma_warp_tile::M_thread) * mma_warp_tile::N_warp + QK_mma.thread_y;
            global_col = tile_x * mma_warp_tile::M_tile + (index % mma_warp_tile::M_thread) * mma_warp_tile::M_warp + QK_mma.thread_x;

            if (global_row >= N || global_col >= M)
                return 0.f;

            bias = has_attn_bias ? attn_bias_[global_row][global_col] : 0.f;

            if (causal && (global_row < (global_col - MN_DIFF)))
                return 0.f;

            return __expf((scale * el + bias) - scale); 
        });

        QK_mma.store_transpose(A_sm.smem);

        __syncthreads();

        // Second matmul:
        V_sm.load(V_, 0, tile_x, M);

        __syncthreads();

        for (int d = 0; d < mma_warp_tile::M_tile; d++) {
            out_mma.mma(A_sm.smem, V_sm.smem, d);
        }

        __syncthreads();
    }

    out_mma.store(O_sm.smem);

    __syncthreads();

    out_mma.store_rowsum(L_, tile_y, N);

    O_sm.store(O_, 0, tile_y, N);
}

// forwards c++ function

std::vector<at::Tensor> flash_cosine_sim_attention_forward(
    torch::Tensor Q,
    torch::Tensor K,
    torch::Tensor V,
    torch::Tensor mask,
    torch::Tensor attn_bias,
    float scale,
    bool causal
) {
    const at::cuda::OptionalCUDAGuard device_guard(device_of(Q));

    const int batch = Q.size(0);
    const int heads = Q.size(1);
    const int N = Q.size(2);
    const int M = K.size(2);
    const int D = Q.size(3);
    const int E = V.size(3);

    auto options = torch::TensorOptions().device(device_of(Q)).dtype(torch::kFloat);

    auto O = at::empty({batch, heads, N, E}, options);
    auto L = at::empty({batch, heads, N}, options);

    const dim3 threads_per_block(256);
    const dim3 blocks(cdiv(N, mma_warp_tile::N_tile), batch * heads);

    const int max_feature_dimension = max(D, E);

    const unsigned shared_mem_size = (mma_warp_tile::N_tile * max_feature_dimension +
                                      mma_warp_tile::M_tile * max_feature_dimension +
                                      mma_warp_tile::N_tile * mma_warp_tile::M_tile) * sizeof(float);

    const bool has_attn_bias = !!attn_bias.numel();
    const bool has_mask = !!mask.numel();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(Q.scalar_type(), "forward_cosine_sim_attention_backward", ([&] {
        forward_kernel<<<blocks, threads_per_block, shared_mem_size>>>(
            ACCESSOR(Q, 4, float),
            ACCESSOR(K, 4, float),
            ACCESSOR(V, 4, float),
            ACCESSOR(O, 4, float),
            ACCESSOR(L, 3, float),
            ACCESSOR(mask, 2, bool),
            ACCESSOR(attn_bias, 3, float),
            scale,
            causal,
            has_mask,
            has_attn_bias
        );
    }));

    // handle error
    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR();

    return { O, L };
}

// backward kernel

// backwards preprocess

// calculate do_scaled = rowsum(do * o)
// done by @ptillet at https://github.com/openai/triton/blob/master/python/tutorials/06-fused-attention.py

template <typename scalar_t>
__global__ void backward_calculate_do_scaled(
    const PackedAccessor<scalar_t, 4> d_out,
    const PackedAccessor<scalar_t, 4> o,
          PackedAccessor<scalar_t, 3> do_scaled
) {
    const int heads = o.size(1);
    const int v_dim = o.size(3);

    const int batch_idx = blockIdx.x / heads;
    const int head_idx = blockIdx.x % heads;
    const int seq_idx = blockIdx.y;
    const int dim_idx = threadIdx.x;

    const int warp_id = threadIdx.x / 32;
    const int lane_id = threadIdx.x & 31;

    const unsigned mask = __ballot_sync(0xFFFFFFFFU, dim_idx < v_dim);

    float val = 0.0f;

    extern __shared__ float _shared_mem_preprocess[];

    float* sm_do_scaled = (float*) &_shared_mem_preprocess;

    auto do_ = d_out[batch_idx][head_idx][seq_idx];
    auto o_ = o[batch_idx][head_idx][seq_idx];
    auto do_scaled_ = do_scaled[batch_idx][head_idx];

    // load into shared memory

    if (dim_idx < v_dim)
        val = do_[dim_idx] * o_[dim_idx];

    // warp shuffle reduce

    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(mask, val, offset);
    }

    if (lane_id == 0)
        sm_do_scaled[warp_id] = val;

    __syncthreads();

    if (warp_id == 0) {
        if (dim_idx < (blockDim.x / 32)) {
            val = sm_do_scaled[lane_id];
        } else{
            val = 0;
        }

        for (int offset = 16; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(mask, val, offset);
        }

        if (dim_idx == 0) {
            do_scaled_[seq_idx] = val;
        }
    }
}

// main backward kernel

template <typename scalar_t>
__global__ void backward_kernel(
    const PackedAccessor<scalar_t, 4> q,
    const PackedAccessor<scalar_t, 4> k,
    const PackedAccessor<scalar_t, 4> v,
    const PackedAccessor<bool, 2> mask,
    const PackedAccessor<scalar_t, 3> attn_bias,
          PackedAccessor<scalar_t, 4> dq,
          PackedAccessor<scalar_t, 4> dk,
          PackedAccessor<scalar_t, 4> dv,
          PackedAccessor<scalar_t, 3> d_attn_bias,
    const PackedAccessor<scalar_t, 4> d_out,
    const PackedAccessor<scalar_t, 3> do_scaled,
    const PackedAccessor<scalar_t, 3> l,
    const float scale,
    const bool causal,
    const bool has_mask,
    const bool has_attn_bias,
    const int row_tile_size,
    const int col_tile_size,
    const int row_tiles,
    const int col_tiles
) {

    const int batch = q.size(0);
    const int head = q.size(1);

    const int batch_idx = blockIdx.x / head;
    const int head_idx = blockIdx.x % head;

    const int q_seq_len = q.size(2);
    const int k_seq_len = k.size(2);
    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    const int num_col_tiles = cdiv(k_seq_len, col_tile_size);
    const int num_row_tiles = cdiv(q_seq_len, row_tile_size);

    const int row_tiles_idx = blockIdx.y / col_tiles;
    const int col_tiles_idx = blockIdx.y % col_tiles;

    const int col_tile_idx = threadIdx.x;
    const int row_tile_idx = threadIdx.y;

    const int lane_id = threadIdx.x & 31;

    const int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
    const int tpb = blockDim.x * blockDim.y;

    const int k_total_el = k_dim * col_tile_size;
    const int v_total_el = v_dim * col_tile_size;

    const int sm_q_offset = row_tile_idx * k_dim;
    const int sm_k_offset = col_tile_idx * k_dim;
    const int sm_v_offset = col_tile_idx * v_dim;
    const int sm_o_offset = row_tile_idx * v_dim;

    auto q_ = q[batch_idx][head_idx];
    auto k_ = k[batch_idx][head_idx];
    auto v_ = v[batch_idx][head_idx];
    auto dq_ = dq[batch_idx][head_idx];
    auto dk_ = dk[batch_idx][head_idx];
    auto dv_ = dv[batch_idx][head_idx];
    auto ds_ = d_attn_bias[head_idx];
    auto do_scaled_ = do_scaled[batch_idx][head_idx];
    auto l_ = l[batch_idx][head_idx];
    auto do_ = d_out[batch_idx][head_idx];
    auto mask_ = mask[batch_idx];

    // handle attention bias

    auto attn_bias_ = has_attn_bias ? attn_bias[head_idx] : attn_bias[0];

    // some variables

    int col_tiles_offset, row_tiles_offset;
    int global_col, global_row;
    bool should_calculate_attn, should_calculate_row, should_calculate_col;

    // shared memory

    extern __shared__ float _shared_mem[];

    float* sm_q = (float*) &_shared_mem;
    float* sm_k = (float*) &sm_q[row_tile_size * k_dim];
    float* sm_v = (float*) &sm_k[col_tile_size * k_dim];
    float* sm_l = (float*) &sm_v[col_tile_size * v_dim];
    float* sm_do_scaled = (float*) &sm_l[row_tile_size];
    float* sm_do = (float*) &sm_do_scaled[row_tile_size];

    // loop

    for (int i = 0; i < num_col_tiles; i++) {
        col_tiles_offset = i * col_tile_size;
        global_col = col_tiles_offset + col_tiles_idx * col_tile_size + col_tile_idx;
        should_calculate_col = global_col < k_seq_len && (!has_mask || mask_[global_col]);

        // coalesced reads
        // cleanup later

        for (
            int offset = 0;
            offset < k_total_el;
            offset += tpb
        ) {
            int sm_idx = offset + thread_idx;
            int gmem_seq_idx = sm_idx / k_dim;
            int gmem_dim_idx = sm_idx % k_dim;

            if (offset < k_total_el)
                sm_k[sm_idx] = k_[col_tiles_offset + col_tiles_idx * col_tile_size + gmem_seq_idx][gmem_dim_idx];
        }

        for (
            int offset = 0;
            offset < v_total_el;
            offset += tpb
        ) {
            int sm_idx = offset + thread_idx;
            int gmem_seq_idx = sm_idx / v_dim;
            int gmem_dim_idx = sm_idx % v_dim;

            if (offset < v_total_el)
                sm_v[sm_idx] = v_[col_tiles_offset + col_tiles_idx * col_tile_size + gmem_seq_idx][gmem_dim_idx];
        }

        for (int j = 0; j < num_row_tiles; j++) {
            row_tiles_offset = j * row_tile_size;
            global_row = row_tiles_offset + row_tiles_idx * row_tile_size + row_tile_idx;
            should_calculate_row = global_row < q_seq_len;

            should_calculate_attn = should_calculate_row &&
                                    should_calculate_col &&
                                    ( !causal ||
                                      (causal && (global_row >= (global_col - k_seq_len + q_seq_len))));

            for (
                int d = col_tile_idx;
                d < k_dim;
                d += col_tile_size
            ) {
                sm_q[row_tile_idx * k_dim + d] = q_[row_tiles_offset + row_tiles_idx * row_tile_size + row_tile_idx][d];
            }

            for (
                int d = col_tile_idx;
                d < v_dim;
                d += col_tile_size
            ) {
                sm_do[row_tile_idx * v_dim + d] = do_[row_tiles_offset + row_tiles_idx * row_tile_size + row_tile_idx][d];
            }

            if (col_tile_idx == 0) {
                sm_do_scaled[row_tile_idx] = do_scaled_[global_row];
                sm_l[row_tile_idx] = l_[global_row];
            }

            __syncthreads();

            float attn = 0;
            float row_sum = 0;
            float dp = 0;

            if (should_calculate_attn) {
                for (int d = 0; d < k_dim; d++) {
                    // dmod is a "hacky" way to avoid bank register conflicts from @ahennequ
                    int dmod = (d + lane_id) % k_dim;
                    attn += sm_q[sm_q_offset + dmod] * sm_k[sm_k_offset + dmod];
                }

                attn *= scale;

                if (has_attn_bias) {
                    attn += attn_bias_[global_row][global_col];
                }

                attn -= scale;
                attn = __expf(attn);

                row_sum = sm_l[row_tile_idx];

                if (row_sum > 1e-8)
                    attn /= row_sum;

                for (int d = 0; d < v_dim; d++) {
                    // accumulate dv to global mem

                    atomicAdd((float*) &dv_[global_col][d], sm_do[sm_o_offset + d] * attn);

                    // calculate dp

                    dp += sm_do[sm_o_offset + d] * sm_v[sm_v_offset + d];
                }
            }

            // calculate dS

            float dS = 0;

            if (should_calculate_attn) {
                float D = sm_do_scaled[row_tile_idx];

                dS = attn * (dp - D);

                if (has_attn_bias) {
                    atomicAdd((float*) &ds_[global_row][global_col], dS);
                }
            }

            __syncthreads();

            // accumulate dq and dk to global mem

            if (should_calculate_attn) {
                dS *= scale;

                for (int d = 0; d < k_dim; d++) {
                    atomicAdd((float*) &dq_[global_row][d], dS * sm_k[sm_k_offset + d]);

                    atomicAdd((float*) &dk_[global_col][d], dS * sm_q[sm_q_offset + d]);
                }
            }

            __syncthreads();
        }
    }
}

// backwards c++ function

std::vector<torch::Tensor> flash_cosine_sim_attention_backward(
    torch::Tensor d_out,
    torch::Tensor o,
    torch::Tensor l,
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    torch::Tensor d_attn_bias,
    torch::Tensor mask,
    torch::Tensor attn_bias,
    float scale,
    bool causal,
    int row_tile_size,
    int col_tile_size,
    int row_tiles,
    int col_tiles
) {
    auto query_device = device_of(q);

    const at::cuda::OptionalCUDAGuard device_guard(query_device);

    const int batch = q.size(0);
    const int heads = q.size(1);
    const int seq   = q.size(2);
    const int k_dim = k.size(3);
    const int v_dim = v.size(3);

    const bool has_attn_bias = !!d_attn_bias.numel();
    const bool has_mask = !!mask.numel();

    auto options = torch::TensorOptions().device(query_device).dtype(torch::kFloat);

    // setup dq, dk, dv

    auto do_scaled = at::empty_like(l, options);

    auto dq = at::zeros_like(q, options);
    auto dk = at::zeros_like(k, options);
    auto dv = at::zeros_like(v, options);

    // setup backwards preprocess call

    const dim3 backwards_preprocess_threads_per_block(next_multiple_of(v_dim, 32));

    const dim3 backwards_preprocess_blocks(batch * heads, seq);

    const unsigned backwards_preprocess_shared_mem_size = cdiv(v_dim, 32) * sizeof(float);

    // setup backwards call

    const dim3 backwards_threads_per_block(col_tile_size, row_tile_size);
    const dim3 backwards_blocks(batch * heads, row_tiles * col_tiles);

    const unsigned backwards_shared_mem_size = (  (row_tile_size + col_tile_size) * k_dim +      // q, k
                                                  (row_tile_size + col_tile_size) * v_dim +      // v, do
                                                  (row_tile_size + col_tile_size)                // l, do_scaled
                                                ) * sizeof(float);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(q.scalar_type(), "forward_cosine_sim_attention_backward", ([&] {
        backward_calculate_do_scaled<scalar_t><<<backwards_preprocess_blocks, backwards_preprocess_threads_per_block, backwards_preprocess_shared_mem_size>>>(
            ACCESSOR(d_out, 4, scalar_t),
            ACCESSOR(o, 4, scalar_t),
            ACCESSOR(do_scaled, 3, scalar_t)
        );

        backward_kernel<scalar_t><<<backwards_blocks, backwards_threads_per_block, backwards_shared_mem_size>>>(
            ACCESSOR(q, 4, scalar_t),
            ACCESSOR(k, 4, scalar_t),
            ACCESSOR(v, 4, scalar_t),
            ACCESSOR(mask, 2, bool),
            ACCESSOR(attn_bias, 3, scalar_t),
            ACCESSOR(dq, 4, scalar_t),
            ACCESSOR(dk, 4, scalar_t),
            ACCESSOR(dv, 4, scalar_t),
            ACCESSOR(d_attn_bias, 3, scalar_t),
            ACCESSOR(d_out, 4, scalar_t),
            ACCESSOR(do_scaled, 3, scalar_t),
            ACCESSOR(l, 3, scalar_t),
            scale,
            causal,
            has_mask,
            has_attn_bias,
            row_tile_size,
            col_tile_size,
            row_tiles,
            col_tiles
        );
    }));

    hipDeviceSynchronize();

    // handle error

    CHECK_LAST_CUDA_ERROR();

    return {dq, dk, dv};
}

// bind

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &flash_cosine_sim_attention_forward, "Flash Cosine-Sim Attention Forward");
    m.def("backward", &flash_cosine_sim_attention_backward, "Flash Cosine-Sim Attention Backward");
}
